#include "hip/hip_runtime.h"
# name of prog cuda, boiler similar to c

#include <stdio.h>
#include <stdlib.h>

#include "../common/book.h"

#define TPB 512
#define N

//be sure to include -arch sm_11 for atomics

// kernel func
__global__ kernel(void){

}

int main(){
  //declar vars
  dim3 gridDim(1,1);
  dim3 blockDim(16,16);

  // allocate some memory

  //init memory

  //exec kernel
  kernel<<<gridDim, blockDim>>>(void);

  //copy mem

  //free mem
  
  return 0;
}
